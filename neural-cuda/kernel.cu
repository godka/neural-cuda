#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#define __CUDA_INTERNAL_COMPILATION__
#include "math_functions.hpp"
#undef __CUDA_INTERNAL_COMPILATION__
#include <stdio.h>
#include "neural-cuda.h"
#define blockMax 500  

__global__ void MatrixMulKernel(const double* A, const double* B, double* C, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N)
		C[i] = A[i] * B[i];
}

__global__ void SigmoidKernel(double* A, double* B, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N)
	{
		B[i] = 1 / (1 + exp(-A[i]));
	}
}
__global__ void DsigmoidKernel(double* A, double* B, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N)
	{
		double a = 1 + exp(-A[i]);
		B[i] = (a - 1) / (a*a);
	}
}

__global__ void ExpKernel(double* A, double* B, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N)
	{
		B[i] = exp(A[i]);
	}
}

HBAPI int MYTHAPI cuda_hadamardProduct(const double *A, const double *B, double *R, unsigned int size)
{
	int blockNum = (size + blockMax - 1) / blockMax;

	MatrixMulKernel << < blockNum, blockMax >> >(A, B, R, size);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	return cudaStatus;
	//}
	return 0;
}

HBAPI int MYTHAPI cuda_dsigmoid(double *A, double *B, unsigned int size)
{
	int blockNum = (size + blockMax - 1) / blockMax;
	DsigmoidKernel << < blockNum, blockMax >> >(A, B, size);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	return 1;
	//}
	return 0;
}

HBAPI int MYTHAPI  cuda_test(){
	printf("hello cuda!\n");
	return 0;
}
HBAPI int MYTHAPI cuda_sigmoid(double *A, double *B, unsigned int size)
{
	int blockNum = (size + blockMax - 1) / blockMax;

	SigmoidKernel << < blockNum, blockMax >> >(A, B, size);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	//cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	return 1;
	//}
	return 0;
}
HBAPI int MYTHAPI cuda_exp(double *A, double *B, unsigned int size)
{
	int blockNum = (size + blockMax - 1) / blockMax;

	SigmoidKernel << < blockNum, blockMax >> >(A, B, size);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return 1;
	}
	return 0;
}